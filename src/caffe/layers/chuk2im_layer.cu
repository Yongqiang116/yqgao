#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2chuk.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void Chuk2imLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  for (int n = 0; n < bottom[0]->num(); ++n) {
    chuk2im_gpu(bottom_data + bottom[0]->offset(n), im_c_, im_h_, im_w_,
        height_, width_, 1, 1,
        height_, width_, top_data + top[0]->offset(n));
  }
}

template <typename Dtype>
void Chuk2imLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  for (int n = 0; n < top[0]->num(); ++n) {
    im2chuk_gpu(top_diff + top[0]->offset(n), im_c_, im_h_, im_w_,
        height_, width_, 1, 1,
        height_, width_, bottom_diff + bottom[0]->offset(n));
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(Chuk2imLayer);

}  // namespace caffe
