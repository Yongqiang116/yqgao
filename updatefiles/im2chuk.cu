#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <stdio.h>

#include "caffe/common.hpp"
#include "caffe/util/im2chuk.hpp"

namespace caffe {

template <typename Dtype>
__global__ void im2chuk_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int channels,
    const int stride_h, const int stride_w,
    const int height_chuk, const int width_chuk,
    Dtype* data_chuk) {
  CUDA_KERNEL_LOOP(index, n) {
    // int num_h_chuk = (height - height_chuk) / stride_h + 1;
    int num_w_chuk = (width - width_chuk) / stride_w + 1;
    int th_width = index / channels % num_w_chuk;
    int th_height = index / channels / num_w_chuk;
    int channel_in = index % channels;
    int height_in = th_height * stride_h;
    int width_in = th_width * stride_w;
    Dtype* data_chuk_ptr = data_chuk;
    data_chuk_ptr += (index * height_chuk + 0) * width_chuk + 0;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + height_in) * width + width_in;
    for (int i = 0; i < height_chuk; ++i) {
      for (int j = 0; j < width_chuk; ++j) {
        *(data_chuk_ptr + i * width_chuk + j) = 
          data_im_ptr[i * width + j];
      }
    }
  }
}

template <typename Dtype>
void im2chuk_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int local_h, const int local_w,
    const int stride_h, const int stride_w,
    Dtype* data_chuk) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  
  int height_chuk = kernel_h + stride_h * (local_h - 1);
  int width_chuk = kernel_w + stride_w * (local_w - 1);
  int num_h_chuk = (height - height_chuk) / stride_h / local_h + 1;
  int num_w_chuk = (width - width_chuk) / stride_w / local_w + 1; 

  int num_kernels = channels * num_h_chuk * num_w_chuk;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2chuk_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, channels,
      stride_h * local_h, stride_w * local_w, 
      height_chuk, width_chuk, data_chuk);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2chuk_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int local_h, const int local_w, const int stride_h, const int stride_w,
    float* data_chuk);
template void im2chuk_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    double* data_chuk);

template <typename Dtype>
__global__ void chuk2im_gpu_kernel(const int n, const Dtype* data_chuk,
    const int height, const int width, const int channels,
    const int stride_h, const int stride_w,
    const int height_chuk, const int width_chuk, 
    const int num_h_chuk, const int num_w_chuk,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width;
    int h = (index / width) % height;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_chuk_start = (w < width_chuk) ? 0 : (w - width_chuk) / stride_w + 1; 
    int w_chuk_end = min(w / stride_w + 1, (width - width_chuk) / stride_w + 1);
    int h_chuk_start = (h < height_chuk) ? 0 : (h - height_chuk) / stride_h + 1; 
    int h_chuk_end = min(h / stride_h + 1, (height - height_chuk) / stride_h + 1);

    for (int h_chuk = h_chuk_start; h_chuk < h_chuk_end; ++h_chuk) {
      for (int w_chuk = w_chuk_start; w_chuk < w_chuk_end; ++w_chuk) {
        int w_out = w - stride_w * w_chuk;
        int h_out = h -  stride_h * h_chuk;
        int c_out = channels * (h_chuk * num_w_chuk + w_chuk) + c;
        val += data_chuk[c_out * height_chuk * width_chuk + h_out * width_chuk + w_out];
      }
    }

    data_im[index] = val;    
  }
}

template <typename Dtype>
void chuk2im_gpu(const Dtype* data_chuk, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int local_h, const int local_w, const int stride_h,
    const int stride_w, Dtype* data_im) {
  int height_chuk = kernel_h + stride_h * (local_h - 1);
  int width_chuk = kernel_w + stride_w * (local_w - 1);
  int num_h_chuk = (height - height_chuk) / stride_h / local_h + 1;
  int num_w_chuk = (width - width_chuk) / stride_w / local_w + 1; 

  int num_kernels = channels * height * width;
  
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  chuk2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_chuk, height, width, channels,
      stride_h * local_h, stride_w * local_w, 
      height_chuk, width_chuk, num_h_chuk, num_w_chuk, 
      data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void chuk2im_gpu<float>(const float* data_chuk, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int local_h, const int local_w, const int stride_h,
    const int stride_w, float* data_im);
template void chuk2im_gpu<double>(const double* data_chuk, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int local_h, const int local_w, const int stride_h,
    const int stride_w, double* data_im);

}  // namespace caffe
