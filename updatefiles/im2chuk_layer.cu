#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2chuk.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void Im2chukLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  for (int n = 0; n < bottom[0]->num(); ++n) {
    im2chuk_gpu(bottom_data + bottom[0]->offset(n), channels_, height_,
        width_, kernel_h_, kernel_w_, local_h_, local_w_,
        stride_h_, stride_w_, top_data + top[0]->offset(n));
  }
}

template <typename Dtype>
void Im2chukLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  for (int n = 0; n < top[0]->num(); ++n) {
    chuk2im_gpu(top_diff + top[0]->offset(n), channels_, height_, width_,
        kernel_h_, kernel_w_, local_h_, local_w_,
        stride_h_, stride_w_, bottom_diff + bottom[0]->offset(n));
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(Im2chukLayer);

}  // namespace caffe
